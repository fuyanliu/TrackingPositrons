#include "hip/hip_runtime.h"
#include <iostream>
#include <thrust/sort.h>

#include "Simulation/PropagateGPU.cuh"
#include "Simulation/TrackGPU.cuh"
#include "Simulation/CudaHelper.hh"

#include "Simulation/BetheEnergyLossCUDA.cuh"

namespace na63 {

/** Forward declarations */
__host__ void PropagateGPU(Simulator* simulator);
__global__ void PropagateKernel(KernelPars args);
__host__ hipError_t DeviceAllocation(Simulator *simulator, KernelPars *p);
__host__ hipError_t DeviceFree(KernelPars *p);

__host__
void PropagateGPU(Simulator* simulator) {

  if (simulator->debug) {
    for (int i=0;i<8;i++) {
      Track t = simulator->GetTrack(i);
      std::cout << t << std::endl;
    }
  }

  if (simulator->debug) std::cout << "Propagating on GPU..." << std::endl;

  // CUDA Parameters
  int N = simulator->TrackSize();
  int threadsPerBlock = 256;
  int blocksPerGrid = (N - 1) / threadsPerBlock + 1;

  // Initialize arguments
  KernelPars kernel_args;
  kernel_args.N = N;
  kernel_args.n_volumes = simulator->geometry->volumes_size();
  kernel_args.tracks = nullptr;
  kernel_args.keys = nullptr;
  kernel_args.materials = nullptr;
  kernel_args.particles = nullptr;
  kernel_args.volume_types = nullptr;
  kernel_args.volumes = nullptr;

  // Allocate memory on device and copy data
  GPUTrack *tracks = simulator->GPUTracks();
  if (simulator->debug) std::cout << "Generated GPU tracks." << std::endl;
  const unsigned size_tracks = N*sizeof(GPUTrack);
  const unsigned size_keys = N*sizeof(int);
  if (CudaError(hipMalloc((void**)&kernel_args.tracks,size_tracks))) return;
  if (CudaError(hipMalloc((void**)&kernel_args.keys,size_keys))) return;
  if (CudaError(hipMemcpy(kernel_args.tracks,tracks,size_tracks,hipMemcpyHostToDevice))) return;
  if (simulator->debug) std::cout << "Copied tracks and keys." << std::endl;
  // Random number generator states
  if (CudaError(hipMalloc(&kernel_args.rng_states,N*sizeof(hiprandState)))) return;
  // Thrust wrappers
  thrust::device_ptr<GPUTrack> devptr_thrust_tracks(kernel_args.tracks);
  thrust::device_ptr<int>      devptr_thrust_keys(kernel_args.keys);
  if (CudaError(DeviceAllocation(simulator, &kernel_args))) return;
  if (simulator->debug) std::cout << "Copied geometry." << std::endl;


  if (simulator->debug) {
    std::cout << "Copied " << N << " tracks of size " << sizeof(GPUTrack) << " bytes each, resulting in a total of " << size_tracks << " bytes of data on the device." << std::endl;
    std::cout << "About to initialize " << blocksPerGrid << " blocks of " << threadsPerBlock << " threads each, resulting in a total of " << blocksPerGrid * threadsPerBlock << " threads." << std::endl;
  }

  int kernel_launches = 0;

  while (1) {

    // Should be dynamic
    kernel_args.steps = 100;
    kernel_args.dl = simulator->step_size;
    kernel_args.rng_seed = kernel_launches;

    // Launch kernel
    kernel_launches++;
    PropagateKernel<<<blocksPerGrid,threadsPerBlock>>>(kernel_args);
    hipDeviceSynchronize();
    thrust::sort_by_key(devptr_thrust_keys, devptr_thrust_keys + N, devptr_thrust_tracks);
    hipDeviceSynchronize();

    // Check if propagation is done
    int done = 0;
    if (CudaError(hipMemcpy((void*)&done,kernel_args.keys,sizeof(int),hipMemcpyDeviceToHost))) return;
    if (done == MAX_INT_VALUE) break;

  } // End kernel launch loop

  if (simulator->debug) std::cout << "Propagation kernel launched "
      << kernel_launches << " times." << std::endl;

  // Copy back and free memory
  if (CudaError(hipMemcpy(tracks,kernel_args.tracks,size_tracks,hipMemcpyDeviceToHost))) return;
  CudaError(DeviceFree(&kernel_args));

  simulator->CopyBackTracks();

  if (simulator->debug)
    for (int i=0;i<8;i++)
      std::cout << simulator->GetTrack(i) << std::endl;

}

/**
 * Frees used memory on the GPU.
 * @return CUDA error code for first failed instruction, otherwise hipSuccess.
 */
__host__
hipError_t DeviceFree(KernelPars *p) {
  hipError_t err;
  if ((err = hipFree(p->tracks))       != hipSuccess) return err;
  if ((err = hipFree(p->keys))         != hipSuccess) return err;
  if ((err = hipFree(p->materials))    != hipSuccess) return err;
  if ((err = hipFree(p->particles))    != hipSuccess) return err;
  if ((err = hipFree(p->volume_types)) != hipSuccess) return err;
  if ((err = hipFree(p->volumes))      != hipSuccess) return err;
  return hipSuccess;
}
/**
 * Attempts to allocate room and copy all geometry information to the device.
 * @return CUDA error code for first failed instruction, otherwise hipSuccess.
 */
__host__
hipError_t DeviceAllocation(Simulator *simulator, KernelPars *p) {
  hipError_t err;

  Geometry *geometry = simulator->geometry;

  // Free previously allocated memory
  if (p->materials != nullptr) {
    if ((err = hipFree(p->materials)) != hipSuccess) return err;
  }
  if (p->particles != nullptr) {
    if ((err = hipFree(p->particles)) != hipSuccess) return err;
  }
  if (p->volume_types != nullptr) {
    if ((err = hipFree(p->volume_types)) != hipSuccess) return err;
  }
  if (p->volumes != nullptr) {
    if ((err = hipFree(p->volumes)) != hipSuccess) return err;
  }

  // Allocate space
  const unsigned size_material = geometry->materials_size()*sizeof(MaterialPars);
  const unsigned size_particle = simulator->particles_size()*sizeof(ParticlePars);
  const unsigned size_volume_type = geometry->volume_types_size()*sizeof(InsideFunction);
  const unsigned size_volume   = geometry->volumes_size()*sizeof(VolumePars);
  err = hipMalloc((void**)&p->materials,
                   size_material);
  if (err != hipSuccess) return err;
  err = hipMalloc((void**)&p->particles,
                   size_particle);
  if (err != hipSuccess) return err;
  err = hipMalloc((void**)&p->volume_types,
                   size_volume_type);
  if (err != hipSuccess) return err;
  err = hipMalloc((void**)&p->volumes,
                   size_volume);
  if (err != hipSuccess) return err;

  // Copy data to device
  err = hipMemcpy(p->materials,
                   geometry->material_arr(),
                   size_material,
                   hipMemcpyHostToDevice);
  if (err != hipSuccess) return err;
  err = hipMemcpy(p->particles,
                   simulator->particle_arr(),
                   size_particle,
                   hipMemcpyHostToDevice);
  if (err != hipSuccess) return err;
  err = hipMemcpy(p->volume_types,
                   geometry->volume_type_arr(),
                   size_volume_type,
                   hipMemcpyHostToDevice);
  if (err != hipSuccess) return err;
  err = hipMemcpy(p->volumes,
                   geometry->volume_arr(),
                   size_volume,
                   hipMemcpyHostToDevice);
  if (err != hipSuccess) return err;

  return hipSuccess;
}

/** Called from host function, lets each thread loop over one particle */
__global__
void PropagateKernel(KernelPars args) {

  // Set and check thread index
  int index = threadIdx.x + blockDim.x * blockIdx.x;
  if (index >= args.N) return;

  // Get memory pointers
  GPUTrack* track = &args.tracks[index];
  if (track->alive == 0) return;
  ParticlePars* particle = &args.particles[track->particle_index];

  // Initialize random number generator
  hiprand_init(args.rng_seed,index,0,&args.rng_states[index]);

  // Perform steps
  for (int i=0;i<args.steps;i++) {

    // Update volume if necessary
    track->volume_index = VolumeQuery(*track,args.volumes,args.n_volumes);
    // If out of bounds or out of energy, break
    if (track->volume_index < 0 || track->momentum[3] < particle->mass) {
      track->alive = 0;
      break;
    }

    __syncthreads();

    // Propagate position
    Step(*track,*particle,args.dl);

    // Run physics
    if (track->volume_index > 0) {
      const int material_index = args.volumes[track->volume_index].material_index;
      if (material_index >= 0) {
        const MaterialPars *material = &args.materials[args.volumes[track->volume_index].material_index];
        __syncthreads();
        CUDA_BetheEnergyLoss(*track,*particle,*material,args.dl,&args.rng_states[index]);
      }
    }

  }

  __syncthreads();

  // Set key to sort by
  int key = MAX_INT_VALUE;
  if (track->alive == 1) {
    key = (int)track->position[0];
  }
  args.keys[index] = key;

}

} // End namespace na63