#include "hip/hip_runtime.h"
#include <iostream>
#include <thrust/sort.h>

#include "Simulation/PropagateGPU.cuh"
#include "Simulation/TrackGPU.cuh"
#include "Simulation/CudaHelper.hh"
#include "Simulation/GetTime.hh"

#include "Simulation/BetheEnergyLossCUDA.cuh"
#include "Simulation/GEANT4BremsstrahlungCUDA.cuh"
#include "Simulation/GEANT4PairProductionCUDA.cuh"
#include "Simulation/DeviceGlobalVariables.cuh"

namespace na63 {

__constant__ VolumePars *volumes;
__constant__ int n_volumes;
__device__ GPUTrack *tracks;
__device__ GPUTrack *track_pool;
__constant__ int maximum_index;
__device__ int *keys;
__device__ Float secondary_threshold;

__constant__ int electron_index;
__constant__ int photon_index;

/** Forward declarations */
__host__ void PropagateGPU(Simulator* simulator);
__global__ void PropagateKernel(KernelPars args,const bool gpu_bremsstrahlung);
__host__ hipError_t DeviceAllocation(Simulator *simulator, KernelPars *p);
__host__ hipError_t DeviceFree(KernelPars *p);

__host__
void PropagateGPU(Simulator* simulator) {

  /*if (simulator->debug) {
    for (int i=0;i<8;i++) {
      Track t = simulator->GetTrack(i);
      std::cout << t << std::endl;
    }
  }*/

  if (simulator->debug) std::cout << "Propagating on GPU..." << std::endl;

  // CUDA Parameters
  int n_tracks_initial = simulator->TrackSize();
  // Use threads equal to the first power of two greater than or equal to the
  // initial amount of tracks.
  int threads = simulator->thread_multiplier*(1 << (int)(log(n_tracks_initial)/log(2)));
  if (threads < n_tracks_initial) threads = threads << 1;
  int threadsPerBlock = 256;
  int blocksPerGrid = (threads - 1) / threadsPerBlock + 1;
  // Allocate more to allow for particle pool
  int N = (1 + simulator->pool_size) * n_tracks_initial;

  // Initialize arguments
  KernelPars kernel_args;
  kernel_args.n_volumes = simulator->geometry->volumes_size();
  kernel_args.materials = nullptr;
  kernel_args.particles = nullptr;
  kernel_args.volume_types = nullptr;
  kernel_args.volumes = nullptr;
  kernel_args.sorting = simulator->sorting;

  // Get track information from simulator
  GPUTrack *tracks_host = new GPUTrack[N];
  simulator->GPUTracks(tracks_host);
  GPUTrack *tracks_device;
  int *keys_host = new int[N];
  // Initialize free tracks
  for (int i=n_tracks_initial;i<N;i++) {
    tracks_host[i].state = FREE;
    keys_host[i] = TRACK_KEY_FREE;
  }
  int *keys_device;
  if (simulator->debug) std::cout << "Generated GPU tracks." << std::endl;

  // Allocate and copy to device
  //const unsigned size_tracks_initial = n_tracks_initial * sizeof(GPUTrack);
  const unsigned size_tracks = N*sizeof(GPUTrack);
  const unsigned size_keys = N*sizeof(int);
  if (simulator->debug) std::cout << "Copying tracks and keys... ";
  if (CudaError(hipMalloc((void**)&tracks_device,size_tracks))) return;
  if (CudaError(hipMalloc((void**)&keys_device,size_keys))) return;
  if (CudaError(hipMemcpy(tracks_device,tracks_host,size_tracks,hipMemcpyHostToDevice))) return;
  if (CudaError(hipMemcpy(keys_device,keys_host,size_keys,hipMemcpyHostToDevice))) return;
  if (simulator->debug) std::cout << N << " track slots copied to device." << std::endl;

  // Random number generator states
  if (CudaError(hipMalloc(&kernel_args.rng_states,N*sizeof(hiprandState)))) return;
  // Thrust wrappers
  thrust::device_ptr<GPUTrack> devptr_thrust_tracks(tracks_device);
  thrust::device_ptr<int>      devptr_thrust_keys(keys_device);
  if (simulator->debug) std::cout << "Copying geometry... ";
  if (CudaError(DeviceAllocation(simulator, &kernel_args))) return;
  if (simulator->debug) std::cout << "OK" << std::endl;

  // Set device global variables
  if (simulator->debug) std::cout << "Copying global variables... ";
  if (CudaError(hipMemcpyToSymbol(HIP_SYMBOL(
    volumes),
    &kernel_args.volumes,
    sizeof(VolumePars*),
    0,
    hipMemcpyHostToDevice
  ))) return;
  if (CudaError(hipMemcpyToSymbol(HIP_SYMBOL(
    tracks),
    &tracks_device,
    sizeof(GPUTrack*),
    0,
    hipMemcpyHostToDevice
  ))) return;
  if (CudaError(hipMemcpyToSymbol(HIP_SYMBOL(
    keys),
    &keys_device,
    sizeof(int*),
    0,
    hipMemcpyHostToDevice
  ))) return;
  int volume_size = simulator->geometry->volumes_size();
  if (CudaError(hipMemcpyToSymbol(HIP_SYMBOL(
    n_volumes),
    &volume_size,
    sizeof(int),
    0,
    hipMemcpyHostToDevice
  ))) return;
  int max_idx = N - 1;
  if (CudaError(hipMemcpyToSymbol(HIP_SYMBOL(
    maximum_index),
    &max_idx,
    sizeof(int),
    0,
    hipMemcpyHostToDevice
  ))) return;
  // Specific particles for processes (Bremsstrahlung)
  int electron_idx = simulator->GetParticleIndex(11);
  if (electron_idx >= 0) {
    if (CudaError(hipMemcpyToSymbol(HIP_SYMBOL(
      electron_index),
      &electron_idx,
      sizeof(int),
      0,
      hipMemcpyHostToDevice
    ))) return;
  }
  int photon_idx = simulator->GetParticleIndex(22);
  if (photon_idx >= 0) {
    if (CudaError(hipMemcpyToSymbol(HIP_SYMBOL(
      photon_index),
      &photon_idx,
      sizeof(int),
      0,
      hipMemcpyHostToDevice
    ))) return;
  }
  if (CudaError(hipMemcpyToSymbol(HIP_SYMBOL(
    secondary_threshold),
    &simulator->secondary_threshold,
    sizeof(Float),
    0,
    hipMemcpyHostToDevice
  ))) return;
  if (simulator->debug) std::cout << "OK" << std::endl;;


  if (simulator->debug) {
    /*std::cout << "Copied " << N << " tracks of size " << sizeof(GPUTrack)
              << " bytes each (of which " << n_tracks_initial 
              << " are initially alive), resulting in a total of " << size_tracks
              << " bytes of data on the device." << std::endl;*/
    std::cout << "About to initialize " << blocksPerGrid << " blocks of "
              << threadsPerBlock << " threads each, resulting in a total of "
              << blocksPerGrid * threadsPerBlock << " threads." << std::endl;
  }

  int kernel_launches = 0;
  bool waitflag = false;

  //////////////////////////////////////////////////////////////////////////////
  //// ACTUAL KERNEL LAUNCH LOOP
  //////////////////////////////////////////////////////////////////////////////
  double elapsed = InSeconds(GetTime());
  while (1) {

    // Should be dynamic
    kernel_args.steps = simulator->steps_per_launch;
    kernel_args.dl = simulator->step_size;
    kernel_args.rng_seed = kernel_launches;

    // Launch kernel
    kernel_launches++;
    if (simulator->debug && kernel_launches % 100 == 0) {
      std::cout << "Kernel launch number "
      << kernel_launches << "..." << std::endl;
    }
    PropagateKernel<<<blocksPerGrid,threadsPerBlock>>>(kernel_args,simulator->gpu_bremsstrahlung);
    hipDeviceSynchronize();
    thrust::sort_by_key(devptr_thrust_keys, devptr_thrust_keys + N, devptr_thrust_tracks);
    hipDeviceSynchronize();

    // Check if propagation is done
    int done = 0;
    if (CudaError(hipMemcpy((void*)&done,keys_device,sizeof(int),hipMemcpyDeviceToHost))) return;
    if (done == TRACK_KEY_DEAD || done == TRACK_KEY_FREE) {
      break;
    } else if (done == TRACK_KEY_WAITING) {
      if (!waitflag) {
        waitflag = true;
      } else {
        std::cout << "Propagation did not finish because of missing memory necessary for generated particles." << std::endl;
        break;
      }
    } else {
      waitflag = false;
    }

    // if (kernel_launches > 1000) {
    //   std::cout << "Timed out at 1000 kernel launches. Possible sorting error?" << std::endl;
    //   break;
    // }

  } // End kernel launch loop
  //////////////////////////////////////////////////////////////////////////////
  elapsed = InSeconds(GetTime()) - elapsed;
  simulator->SetBenchmark(elapsed);
  std::cout << "Propagation ran for " << elapsed << " seconds." << std::endl;

  if (simulator->debug) std::cout << "Propagation kernel launched "
      << kernel_launches << " times." << std::endl;

  // Copy back and free memory
  if (CudaError(hipMemcpy(tracks_host,tracks_device,size_tracks,hipMemcpyDeviceToHost))) return;
  CudaError(DeviceFree(&kernel_args));

  simulator->CopyBackTracks(tracks_host,N);
  delete tracks_host;
  delete keys_host;

  // if (CudaError(hipMemcpy(keys_host,keys_device,size_keys,hipMemcpyDeviceToHost))) return;
  // hipDeviceSynchronize();
  // for (int i=0;i<N;i++) {
  //   std::cout << tracks_host[i].state << ", " << keys_host[i] << std::endl;
  // }

  CudaError(hipFree(keys_device));
  CudaError(hipFree(tracks_device));
  CudaError(hipFree(kernel_args.rng_states));

  /*if (simulator->debug)
    for (int i=0;i<8;i++)
      std::cout << simulator->GetTrack(i) << std::endl;*/

}

/**
 * Frees used memory on the GPU.
 * @return CUDA error code for first failed instruction, otherwise hipSuccess.
 */
__host__
hipError_t DeviceFree(KernelPars *p) {
  hipError_t err;
  if ((err = hipFree(p->materials))    != hipSuccess) return err;
  if ((err = hipFree(p->particles))    != hipSuccess) return err;
  if ((err = hipFree(p->volume_types)) != hipSuccess) return err;
  if ((err = hipFree(p->volumes))      != hipSuccess) return err;
  return hipSuccess;
}
/**
 * Attempts to allocate room and copy all geometry information to the device.
 * @return CUDA error code for first failed instruction, otherwise hipSuccess.
 */
__host__
hipError_t DeviceAllocation(Simulator *simulator, KernelPars *p) {
  hipError_t err;

  Geometry *geometry = simulator->geometry;

  // Free previously allocated memory
  if (p->materials != nullptr) {
    if ((err = hipFree(p->materials)) != hipSuccess) return err;
  }
  if (p->particles != nullptr) {
    if ((err = hipFree(p->particles)) != hipSuccess) return err;
  }
  if (p->volume_types != nullptr) {
    if ((err = hipFree(p->volume_types)) != hipSuccess) return err;
  }
  if (p->volumes != nullptr) {
    if ((err = hipFree(p->volumes)) != hipSuccess) return err;
  }

  // Allocate space
  const unsigned size_material = geometry->materials_size()*sizeof(MaterialPars);
  const unsigned size_particle = simulator->particles_size()*sizeof(ParticlePars);
  const unsigned size_volume_type = geometry->volume_types_size()*sizeof(InsideFunction);
  const unsigned size_volume   = geometry->volumes_size()*sizeof(VolumePars);
  err = hipMalloc((void**)&p->materials,
                   size_material);
  if (err != hipSuccess) return err;
  err = hipMalloc((void**)&p->particles,
                   size_particle);
  if (err != hipSuccess) return err;
  err = hipMalloc((void**)&p->volume_types,
                   size_volume_type);
  if (err != hipSuccess) return err;
  err = hipMalloc((void**)&p->volumes,
                   size_volume);
  if (err != hipSuccess) return err;

  // Copy data to device
  err = hipMemcpy(p->materials,
                   geometry->material_arr(),
                   size_material,
                   hipMemcpyHostToDevice);
  if (err != hipSuccess) return err;
  err = hipMemcpy(p->particles,
                   simulator->particle_arr(),
                   size_particle,
                   hipMemcpyHostToDevice);
  if (err != hipSuccess) return err;
  err = hipMemcpy(p->volume_types,
                   geometry->volume_type_arr(),
                   size_volume_type,
                   hipMemcpyHostToDevice);
  if (err != hipSuccess) return err;
  err = hipMemcpy(p->volumes,
                   geometry->volume_arr(),
                   size_volume,
                   hipMemcpyHostToDevice);
  if (err != hipSuccess) return err;

  return hipSuccess;
}

/** Called from host function, lets each thread loop over one particle */
__global__
void PropagateKernel(KernelPars args, const bool gpu_bremsstrahlung) {

  // Set and check thread index
  int index = threadIdx.x + blockDim.x * blockIdx.x;
  if (index > maximum_index) return;
  
  // Get memory pointers
  GPUTrack* track = &tracks[index];
  // printf("Index %i has state %i and key %i\n",index,track->state,keys[index]);
  // printf("Index %i, key %i, state %i\n",index,keys[index],track->state);

  // if (index < 10) printf("Track %i, state %i, position (%g,%g,%g)\n",index,track->state,track->position[0],track->position[1],track->position[2]);

  if (track->state == WAITING) {
    track->state = ALIVE;
  } else {
    if (track->state != ALIVE) {
      return;
    }
  }
  ParticlePars* particle = &args.particles[track->particle_index];
  hiprandState* rng_state = &args.rng_states[index];

  __syncthreads();

  // Initialize random number generator
  hiprand_init(args.rng_seed,index,0,rng_state);

  // Perform steps
  for (int i=0;i<args.steps;i++) {

    // Run physics
    if (track->volume_index > 0) {
      const int material_index = args.volumes[track->volume_index].material_index;
      if (material_index >= 0) {
        const MaterialPars *material = &args.materials[args.volumes[track->volume_index].material_index];
        if (gpu_bremsstrahlung) {
          CUDA_GEANT4Bremsstrahlung(
            track,
            particle,
            material,
            args.dl,
            rng_state, 
            index
          );
          if (track->state != ALIVE) return;
          __syncthreads();
          CUDA_GEANT4PairProduction(
            track,
            particle,
            material,
            args.dl,
            rng_state, 
            index
          );
          if (track->state != ALIVE) return;
          __syncthreads();
        }
        CUDA_BetheEnergyLoss(
          *track,
          *particle,
          *material,
          args.dl,
          index,
          rng_state
        );
        if (track->state != ALIVE) return;
      }
    }

    // Update volume if necessary
    track->volume_index = VolumeQuery(*track);
    // If out of bounds or out of energy, kill and break
    if (track->volume_index < 0) {
      UpdateState(index,DEAD);
      break;
    }

    __syncthreads();

    // Propagate position
    Step(*track,*particle,args.dl);

  }

  __syncthreads();

  // Set key to sort by
  // if (track->state == WAITING) {
  //   key = TRACK_KEY_WAITING;
  // } else if (track->state == ALIVE) {

  // IMPORTANT that non-alive tracks don't get sorted!!
  if (track->state != ALIVE) return;

  int key;
  if (args.sorting < 3) {
    key = (int)track->position[args.sorting];
  } else if (args.sorting == RADIUS) { // if (args.sorting == RADIUS) {
    // Radius sorting
    key = ThreeVector_Length(track->position);
  } else {
    key = track->particle_id;
  }
  keys[index] = key;

  //printf("%i, %i, (%f,%f,%f,%f), (%f,%f,%f,%f)\n",key,track->particle_id,track->position[0],track->position[1],track->position[2],track->position[3],track->momentum[0],track->momentum[1],track->momentum[2],track->momentum[3]);

}

} // End namespace na63