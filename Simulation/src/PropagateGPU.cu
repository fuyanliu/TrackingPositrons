#include "hip/hip_runtime.h"
#include <iostream>
#include <thrust/sort.h>

#include "Simulation/PropagateGPU.cuh"
#include "Simulation/TrackGPU.cuh"
#include "Simulation/CudaHelper.hh"

#include "Simulation/BetheEnergyLossCUDA.cuh"

namespace na63 {

/** Forward declarations */
__host__ void PropagateGPU(Simulator* simulator);
__global__ void PropagateKernel(KernelPars args);
__host__ hipError_t DeviceAllocation(Simulator *simulator, KernelPars *p);
__host__ hipError_t DeviceFree(KernelPars *p);

__host__
void PropagateGPU(Simulator* simulator) {

  /*if (simulator->debug) {
    for (int i=0;i<8;i++) {
      Track t = simulator->GetTrack(i);
      std::cout << t << std::endl;
    }
  }*/

  if (simulator->debug) std::cout << "Propagating on GPU..." << std::endl;

  // CUDA Parameters
  int n_tracks_initial = simulator->TrackSize();
  // Use threads equal to the first power of two greater than or equal to the
  // initial amount of tracks.
  int threads = 1 << (int)(log(n_tracks_initial)/log(2));
  if (threads < n_tracks_initial) threads = threads << 1;
  int threadsPerBlock = 256;
  int blocksPerGrid = (threads - 1) / threadsPerBlock + 1;
  // Actually allocate three times as much to allow for pair production
  int N = 3 * n_tracks_initial;

  // Initialize arguments
  KernelPars kernel_args;
  kernel_args.N = N;
  kernel_args.n_volumes = simulator->geometry->volumes_size();
  kernel_args.tracks = nullptr;
  kernel_args.keys = nullptr;
  kernel_args.materials = nullptr;
  kernel_args.particles = nullptr;
  kernel_args.volume_types = nullptr;
  kernel_args.volumes = nullptr;

  // Get track information from simulator
  GPUTrack *tracks = new GPUTrack[N];
  simulator->GPUTracks(tracks);
  // Initialize free tracks
  for (int i=n_tracks_initial;i<N;i++) {
    tracks[i].state = STATE_FREE;
  }
  if (simulator->debug) std::cout << "Generated GPU tracks." << std::endl;

  // Allocate and copy to device
  const unsigned size_tracks_initial = n_tracks_initial * sizeof(GPUTrack);
  const unsigned size_tracks = N*sizeof(GPUTrack);
  const unsigned size_keys = N*sizeof(int);
  if (CudaError(hipMalloc((void**)&kernel_args.tracks,size_tracks))) return;
  if (CudaError(hipMalloc((void**)&kernel_args.keys,size_keys))) return;
  if (CudaError(hipMemcpy(kernel_args.tracks,tracks,size_tracks_initial,hipMemcpyHostToDevice))) return;
  if (simulator->debug) std::cout << "Copied tracks and keys." << std::endl;
  // Random number generator states
  if (CudaError(hipMalloc(&kernel_args.rng_states,N*sizeof(hiprandState)))) return;
  // Thrust wrappers
  thrust::device_ptr<GPUTrack> devptr_thrust_tracks(kernel_args.tracks);
  thrust::device_ptr<int>      devptr_thrust_keys(kernel_args.keys);
  if (CudaError(DeviceAllocation(simulator, &kernel_args))) return;
  if (simulator->debug) std::cout << "Copied geometry." << std::endl;


  if (simulator->debug) {
    std::cout << "Copied " << N << " tracks of size " << sizeof(GPUTrack)
              << " bytes each (of which " << n_tracks_initial 
              << " are initially alive), resulting in a total of " << size_tracks
              << " bytes of data on the device." << std::endl;
    std::cout << "About to initialize " << blocksPerGrid << " blocks of "
              << threadsPerBlock << " threads each, resulting in a total of "
              << blocksPerGrid * threadsPerBlock << " threads." << std::endl;
  }

  int kernel_launches = 0;

  while (1) {

    // Should be dynamic
    kernel_args.steps = 100;
    kernel_args.dl = simulator->step_size;
    kernel_args.rng_seed = kernel_launches;

    // Launch kernel
    kernel_launches++;
    PropagateKernel<<<blocksPerGrid,threadsPerBlock>>>(kernel_args);
    hipDeviceSynchronize();
    thrust::sort_by_key(devptr_thrust_keys, devptr_thrust_keys + N, devptr_thrust_tracks);
    hipDeviceSynchronize();

    // Check if propagation is done
    int done = 0;
    if (CudaError(hipMemcpy((void*)&done,kernel_args.keys,sizeof(int),hipMemcpyDeviceToHost))) return;
    if (done == TRACK_KEY_DEAD) break;

  } // End kernel launch loop

  if (simulator->debug) std::cout << "Propagation kernel launched "
      << kernel_launches << " times." << std::endl;

  // Copy back and free memory
  if (CudaError(hipMemcpy(tracks,kernel_args.tracks,size_tracks,hipMemcpyDeviceToHost))) return;
  CudaError(DeviceFree(&kernel_args));

  simulator->CopyBackTracks(tracks,N);

  /*if (simulator->debug)
    for (int i=0;i<8;i++)
      std::cout << simulator->GetTrack(i) << std::endl;*/

}

/**
 * Frees used memory on the GPU.
 * @return CUDA error code for first failed instruction, otherwise hipSuccess.
 */
__host__
hipError_t DeviceFree(KernelPars *p) {
  hipError_t err;
  if ((err = hipFree(p->tracks))       != hipSuccess) return err;
  if ((err = hipFree(p->keys))         != hipSuccess) return err;
  if ((err = hipFree(p->materials))    != hipSuccess) return err;
  if ((err = hipFree(p->particles))    != hipSuccess) return err;
  if ((err = hipFree(p->volume_types)) != hipSuccess) return err;
  if ((err = hipFree(p->volumes))      != hipSuccess) return err;
  return hipSuccess;
}
/**
 * Attempts to allocate room and copy all geometry information to the device.
 * @return CUDA error code for first failed instruction, otherwise hipSuccess.
 */
__host__
hipError_t DeviceAllocation(Simulator *simulator, KernelPars *p) {
  hipError_t err;

  Geometry *geometry = simulator->geometry;

  // Free previously allocated memory
  if (p->materials != nullptr) {
    if ((err = hipFree(p->materials)) != hipSuccess) return err;
  }
  if (p->particles != nullptr) {
    if ((err = hipFree(p->particles)) != hipSuccess) return err;
  }
  if (p->volume_types != nullptr) {
    if ((err = hipFree(p->volume_types)) != hipSuccess) return err;
  }
  if (p->volumes != nullptr) {
    if ((err = hipFree(p->volumes)) != hipSuccess) return err;
  }

  // Allocate space
  const unsigned size_material = geometry->materials_size()*sizeof(MaterialPars);
  const unsigned size_particle = simulator->particles_size()*sizeof(ParticlePars);
  const unsigned size_volume_type = geometry->volume_types_size()*sizeof(InsideFunction);
  const unsigned size_volume   = geometry->volumes_size()*sizeof(VolumePars);
  err = hipMalloc((void**)&p->materials,
                   size_material);
  if (err != hipSuccess) return err;
  err = hipMalloc((void**)&p->particles,
                   size_particle);
  if (err != hipSuccess) return err;
  err = hipMalloc((void**)&p->volume_types,
                   size_volume_type);
  if (err != hipSuccess) return err;
  err = hipMalloc((void**)&p->volumes,
                   size_volume);
  if (err != hipSuccess) return err;

  // Copy data to device
  err = hipMemcpy(p->materials,
                   geometry->material_arr(),
                   size_material,
                   hipMemcpyHostToDevice);
  if (err != hipSuccess) return err;
  err = hipMemcpy(p->particles,
                   simulator->particle_arr(),
                   size_particle,
                   hipMemcpyHostToDevice);
  if (err != hipSuccess) return err;
  err = hipMemcpy(p->volume_types,
                   geometry->volume_type_arr(),
                   size_volume_type,
                   hipMemcpyHostToDevice);
  if (err != hipSuccess) return err;
  err = hipMemcpy(p->volumes,
                   geometry->volume_arr(),
                   size_volume,
                   hipMemcpyHostToDevice);
  if (err != hipSuccess) return err;

  return hipSuccess;
}

/** Called from host function, lets each thread loop over one particle */
__global__
void PropagateKernel(KernelPars args) {

  // Set and check thread index
  int index = threadIdx.x + blockDim.x * blockIdx.x;
  if (index >= args.N) return;

  // Get memory pointers
  GPUTrack* track = &args.tracks[index];
  if (track->state != STATE_ALIVE) return;
  ParticlePars* particle = &args.particles[track->particle_index];

  // Initialize random number generator
  hiprand_init(args.rng_seed,index,0,&args.rng_states[index]);

  // Perform steps
  for (int i=0;i<args.steps;i++) {

    // Update volume if necessary
    track->volume_index = VolumeQuery(*track,args.volumes,args.n_volumes);
    // If out of bounds or out of energy, break
    if (track->volume_index < 0 || track->momentum[3] < particle->mass) {
      track->state = STATE_DEAD;
      break;
    }

    __syncthreads();

    // Propagate position
    Step(*track,*particle,args.dl);

    // Run physics
    if (track->volume_index > 0) {
      const int material_index = args.volumes[track->volume_index].material_index;
      if (material_index >= 0) {
        const MaterialPars *material = &args.materials[args.volumes[track->volume_index].material_index];
        __syncthreads();
        CUDA_BetheEnergyLoss(*track,*particle,*material,args.dl,&args.rng_states[index]);
      }
    }

  }

  __syncthreads();

  // Set key to sort by
  int key = TRACK_KEY_DEAD;
  if (track->state == STATE_ALIVE) {
    key = (int)track->position[0];
  }
  args.keys[index] = key;

}

} // End namespace na63