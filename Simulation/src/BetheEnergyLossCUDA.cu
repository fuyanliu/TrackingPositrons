#include "Simulation/BetheEnergyLossCUDA.cuh"
#include "Simulation/TrackGPU.cuh"
#include "Simulation/Landau.cuh"
#include "Geometry/LibraryCUDA.cuh"

// K/A = 0.307075 MeV g^-1 cm^2

namespace na63 {

__device__
void CUDA_BetheEnergyLoss(GPUTrack& track, const ParticlePars& particle,
    const MaterialPars& material, const Float dl, hiprandState *rng_state) {

  // Only treat particles with charge
  if (track.charge == 0) return;

  Float mass = particle.mass;

  // Don't handle electrons for now
  if (mass < 1 * MeV) return;

  // Get -<dE/dx> and sigma
  LandauParameters p = CUDA_GetSkewedLandauParameters(
      CUDA_Beta(track.momentum[3],mass),mass,
      material.atomic_number,material.mean_excitation_potential,dl);

  // Get random number from Landau distribution
  Float energy_loss = ThrowLandau(p.mean,p.sigma,hiprand_uniform(rng_state));

  // Update track
  CUDA_UpdateEnergy(track.momentum,mass,-energy_loss * dl);
}

} // End namespace na63