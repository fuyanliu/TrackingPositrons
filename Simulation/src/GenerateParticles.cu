#include "hip/hip_runtime.h"
#include <iostream>

#include "Simulation/GenerateParticles.h"
#include "Simulation/CudaHelper.h"

namespace Simulation {

  __global__
  void cuda_generateParticles_electrons(simple_particle_t* p_, const int N) {
    int index = threadIdx.x + blockDim.x * blockIdx.x;
    if (index >= N) return;
    float px = -0.5e-2 + 1e-2 * (index + 1) / N;
    float py =  0.5e-2 - 1e-2 * (index + 1) / N;
    float pz =  0.1 + 9.9 * (index + 1) / N;
    simple_particle_t* p = &p_[index];
    p->p[0] = px;
    p->p[1] = py;
    p->p[2] = pz;
    p->r[0] = 0;
    p->r[1] = 0;
    p->r[2] = 0;
    p->m = 5.109989e-4;
    p->q = -1;
  }

  __host__
  void generateParticles(simple_particle_t* p, simulator_args_t args) {
    const int N = args.nParticles;
    int threadsPerBlock = 256;
    int blocksPerGrid = (N - 1) / threadsPerBlock + 1;
    const int dataSize = N*sizeof(simple_particle_t);

    simple_particle_t* devicePtr = NULL;
    
    if (error(hipMalloc((void**)&devicePtr,dataSize))) return;
    if (error(hipMemcpy((void*)devicePtr,p,dataSize,hipMemcpyHostToDevice))) return;

    if (args.debug) {
      std::cout << "Copied " << N << " instances of size " << sizeof(simple_particle_t) << " bytes each, resulting in a total of " << dataSize << " bytes of data on the device." << std::endl;
      std::cout << "About to initialize " << blocksPerGrid << " blocks of " << threadsPerBlock << " each, resulting in a total of " << blocksPerGrid * threadsPerBlock << " threads." << std::endl;
    }
    
    cuda_generateParticles_electrons<<<blocksPerGrid,threadsPerBlock>>>(devicePtr,N);
    hipDeviceSynchronize();

    if (error(hipMemcpy(p,devicePtr,dataSize,hipMemcpyDeviceToHost))) return;
    if (error(hipFree(devicePtr))) return;
  }

}