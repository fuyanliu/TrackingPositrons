#include "hip/hip_runtime.h"
#include <stdio.h>

#include "Simulation/GenerateParticles.h"

__global__ void cudaPopulateElectrons(simple_particle_t* _p, const int N) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index > N) return;
  float px = -0.05 + 0.1 * (N - index) / index;
  float py =  0.05 - 0.1 * (N - index) / index;
  float pz =  0.10 + 9.9 * (N - index) / index;
  simple_particle_t *p = &_p[index];
  p->p[0] = px;
  p->p[1] = py;
  p->p[2] = pz;
  p->r[0] = 0;
  p->r[1] = 0;
  p->r[2] = 0;
  p->m = 1;
  p->q = -1;
}

int error(hipError_t err) {
  if (err == hipSuccess) return 0;
  printf("An error occurred.\n");
  return -1;
}

void GenerateParticles(simple_particle_t* p, const int N) {
  int threadsPerBlock = 256;
  int blocksPerGrid = (N - 1) / threadsPerBlock + 1;
  const int dataSize = N*sizeof(simple_particle_t);

  void* devicePtr = (void*)p;
  
  if (error(hipMalloc(&devicePtr,dataSize))) return;
  if (error(hipMemcpy(devicePtr,p,dataSize,hipMemcpyHostToDevice))) return;

  cudaPopulateElectrons<<<blocksPerGrid,threadsPerBlock>>>(p,N);

  if (error(hipMemcpy(p,devicePtr,dataSize,hipMemcpyDeviceToHost))) return;
  if (error(hipFree(devicePtr))) return;
}