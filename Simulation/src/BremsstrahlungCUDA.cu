#include "hip/hip_runtime.h"
#include "Simulation/BremsstrahlungCUDA.cuh"
#include "Geometry/Constants.hh"

namespace na63 {

__device__
inline void CUDA_BremsstrahlungPhoton(GPUTrack& mother,
    const MaterialPars& material, const ParticlePars& particle,
    const Float dl, hiprandState *rng_state) {

  // Must have sufficient energy to be considered
  if (mother.momentum[3] <= 2*kElectronMass) return;

  // See if anything happens
  Float chance_to_interact = dl / material.radiation_length;
  if (hiprand_uniform(rng_state) > chance_to_interact) return;

  // Get electron/positron energy and momentum
  Float child_energy = mother.momentum[3] / 2;
  Float child_momentum = sqrt(child_energy*child_energy - kElectronMass*kElectronMass);
  Float phi = kPi * hiprand_uniform(rng_state);
  Float theta = 2 * kPi * hiprand_uniform(rng_state);
  GPUThreeVector direction;
  CUDA_SphericalToCartesian(direction,child_momentum,phi,theta);

  // Spawn electron and positron back to back and boost them
  GPUTrack electron;
  electron.particle_id = 11;
  electron.charge = -1;
  FourVector_Copy(electron.position,mother.position);
  electron.momentum[0] = direction[0];
  electron.momentum[1] = direction[1];
  electron.momentum[2] = direction[2];
  electron.momentum[3] = child_energy;
  GPUTrack positron = electron;
  positron.charge = 1;
  ThreeVector_Negate(positron.momentum,positron.momentum);


  // Queue new particles for propagation and murder photon
  mother.state = STATE_DEAD;
  // ...

}

__device__
inline void CUDA_BremsstrahlungElectron(GPUTrack& mother,
    const MaterialPars& material, const ParticlePars& particle,
    const Float dl, hiprandState *rng_state) {

  // See if anything happens
  Float chance_to_interact = dl / material.radiation_length;
  if (hiprand_uniform(rng_state) > chance_to_interact) return;

  // Distribute energy from 10% to 30% of electron energy
  Float photon_energy = mother.momentum[3] * (0.1 + 0.2 * hiprand_uniform(rng_state));

  // Photon direction
  Float mother_phi = CUDA_CartesianToSpherical_Phi(mother.momentum[0],mother.momentum[1]);
  Float mother_theta = CUDA_CartesianToSpherical_Theta(mother.momentum[0],mother.momentum[1],mother.momentum[2]);
  Float mother_gamma = CUDA_Gamma(mother.momentum[3],particle.mass);
  Float phi = mother_phi + mother.charge * asin(1 / mother_gamma);
  GPUThreeVector photon_direction;
  CUDA_SphericalToCartesian(photon_direction,photon_energy,mother_theta,phi);

  // Create new track
  GPUTrack photon;
  photon.particle_id = 22;
  FourVector_Copy(photon.position,mother.position);
  ThreeVector_Copy(photon.momentum,photon_direction);
  photon.momentum[3] = photon_energy;

  // Subtract from mother
  ThreeVector_Subtract(mother.momentum,photon.momentum,mother.momentum);
  
  // Spawn photon
  // ...

}

__device__
void CUDA_Bremsstrahlung(GPUTrack& mother, const MaterialPars& material,
    const ParticlePars& particle, const Float dl, hiprandState *rng_state) {

  if (mother.particle_id == 11) CUDA_BremsstrahlungElectron(mother,material,particle,dl,rng_state);
  if (mother.particle_id == 22) CUDA_BremsstrahlungPhoton(mother,material,particle,dl,rng_state);

}

} // End namespace na63